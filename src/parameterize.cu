#include "hip/hip_runtime.h"
#include "parameterize.h"

void getXY( std::vector<Event>& events, 
            thrust::device_vector<int>& x,
            thrust::device_vector<int>& y)
    {
        for(int i = 0; i < events.size(); i++)
        {
            x[i] = events[i].x;
            y[i] = events[i].y;
        }
    }

struct minus
{
    __host__ __device__
        int operator()(const int &a, const int &b)const{
            return a - b;
        }
};

void getMinux(thrust::device_vector<int>& src,
                 thrust::device_vector<int>& dst)
    {
        thrust::transform(src.begin()+1, src.end(), src.begin(),dst.begin(), minus());
    }

struct stepp
{
    __host__ __device__
        double operator()(const int &x_m, const int &y_m)const{
            return (double)(x_m*x_m + y_m*y_m);
        }
};

void getStep(thrust::device_vector<int>& x, 
                thrust::device_vector<int>& y,
                thrust::device_vector<double>& step)
    {
        thrust::transform(x.begin(), x.end(), y.begin(), step.begin(), stepp());
        for(int i = 0; i < step.size(); i++)
        {
            step[i] = sqrt(step[i]);
        }
        thrust::exclusive_scan(step.begin(), step.end(), step.begin());
    }

void test(std::vector<Event>& events,
            std::vector<int>& x, 
            std::vector<int>& y
            )
    {
        thrust::device_vector<int> _x(x.begin(), x.end());
        thrust::device_vector<int> _y(y.begin(), y.end());
        thrust::device_vector<double> step(events.size()-1);
        getXY(events, _x, _y);
        thrust::copy(_x.begin(), _x.end(), x.begin());
        thrust::copy(_y.begin(), _y.end(), y.begin());
        std::cout<<x[1]-x[0]<<std::endl;
        std::cout<<x[2]-x[1]<<std::endl;
        getMinux(_x, _x);
        getStep(_x, _y, step);
        std::cout<<sqrt(_x[0]*_x[0] + _y[0]*_y[0])<<std::endl;
        std::cout<<step[0]<<" "<<step[1]<<std::endl;
    }


void spline(thrust::device_vector<double>& x, 
            thrust::device_vector<int>& y,
            double bound1, 
            double boundn,
            parameter &p )
    {
        //总点数
        int n  = x.size();
        // h[i] = x[i+1] - x[i]
        thrust::device_vector<double> h(n - 1);
        thrust::transform(x.begin() + 1, x.end(), x.begin(), h.begin(), thrust::minus<double>());
        // 稀疏矩阵
        thrust::device_vector<double> a(n, 2);   // 中间一斜列, n个
        thrust::device_vector<double> c(n);      // 上方一斜列 n-1个
        c[0] = 1;
        for(int i = 1; i < n - 1; i++)
            c[i] = h[i] / (h[i-1] + h[i]);
        thrust::device_vector<double> b(n);      // 下方一斜列 n-1个
        for(int i = 1; i < n - 1; i++)
            b[i] = h[i-1] / (h[i-1] + h[i]);
        b[n-1] = 1; 
        b[0] = 0; //b[0]实际上不存在
        c[n-1] = 0; //c[n-1]实际上不存在

        // 右边的d
        thrust::device_vector<double> d(n);
        d[0] = 6*((y[1] - y[0])/h[0] - bound1)/h[0];
        d[n - 1] = 6*(boundn - (y[n-1] - y[n-2])/h[n-2])/h[n-2];
        for(int i = 1; i < n-1; i++)
            d[i] = 6*((y[i+1]-y[i])/h[i] - (y[i] - y[i-1])/h[i-1])/(h[i]+h[i-1]);

        // 追赶法解方程组
        thrust::device_vector<double> l(n); 
        thrust::device_vector<double> m(n);
        thrust::device_vector<double> u(n);
        l[0] = a[0];
        u[0] = c[0]/l[0];
        m[0] = 0;   //实际上不存在
        for(int i = 1; i < n; i++)
        {
            l[i] = a[i] - b[i]*u[i-1];
            u[i] = c[i]/l[i];
            m[i] = b[i];
        }
        //AM=d => LUM=Lt=d, 先解决Lt=d
        thrust::device_vector<double> t(n);
        t[0] = d[0]/l[0];
        for(int i = 1; i < n; i++)
            t[i] = d[i] - m[i]*t[i-1];
        
        //解决t=UM, M就是每个转折点的二阶导数
        thrust::device_vector<double> M(n);
        M[n-1] = t[n-1];
        for(int i = n-2; i >=0; i--)
            M[i] = t[i] - u[i]*M[i+1];
        
        double A[n-1], B[n-1], C[n-1], D[n-1];
        for(int i = 0; i < n-1; i++)
        {
            A[i] = y[i];
            B[i] = (y[i+1]-y[i])/h[i] - h[i]*M[i]/2 - h[i]*(M[i+1] - M[i])/6;
            C[i] = M[i]/2;
            D[i] = (M[i+1] - M[i])/h[i]/6;
        }
        double s[n];
        thrust::copy(x.begin(), x.end(),s);
        std::cout<<"reconstruct start"<<std::endl;
        p.reconstruct(n, s, A, B, C, D);
        p.paramWriter("/home/zw/rotate/param.txt");
        std::cout<<"reconstruct success"<<std::endl;
    }

void splint(parameter& p, double &x, double &y)
    {
        //二分法查找x点所在区间
        int start, end, mid;
        int size = p.getSize();
        double *S, *A, *B, *C, *D;
        S = new double[size];
        A = new double[size -1];
        B = new double[size -1];
        C = new double[size -1];
        D = new double[size -1];
        p.getParam(S, A, B, C, D);
        start = 0; end = size - 1;

        while (end - start > 1)
        {
            mid = (end + start)/2;
            if(S[mid] > x) end = mid;
            else start = mid;
        }
        int index = S[mid] > x?(mid-1):mid;
        y = A[index] + B[index]*(x - S[index])+C[index]*(x-S[index])*(x-S[index])+D[index]*(x-S[index])*(x-S[index])*(x-S[index]);        
        delete S;
        delete A;
        delete B;
        delete C;
        delete D;
    }

void getLineXY(std::vector<Event> &events, parameter &px, parameter &py)
{
    int n = events.size();
    thrust::device_vector<int> x(n);
    thrust::device_vector<int> y(n);
    getXY(events, x, y);
    thrust::device_vector<double> step(n);
    getStep(x, y, step);
    spline(step, x, 0, 0, px);
    spline(step, y, 0, 0, py);
}

template <typename T>
struct triple
{
    __host__ __device__
        T operator()(const T& x) const { 
            return x * x * x;
        }
};

template <typename T>
struct power
{
    int time;

    power(int _time) : time(_time){}

    __host__ __device__
        T operator()(const T& x) const{
            T temp = 1;
            for(int i = 0; i < time; i++)
            {
                temp *= x;
            }
            return temp;
        }
};


struct power_xy
{
    int time;

    power_xy(int _time) : time(_time){}

    __host__ __device__
        double operator()(const double& x, const int& y) const{
            double temp = 1.0;
            for(int i = 0; i < time; i++)
            {
                temp *= x;
            }
            return temp * y;
            
        }
};

void least_square(thrust::device_vector<double>& x, 
                    thrust::device_vector<int>& y)
    {
        int n = x.size();
        double _x[7], _xy[4];
        for(int i = 0; i < 7; i++)
        {
            _x[i] = thrust::transform_reduce(x.begin(), x.end(), power<double>(i), 0, thrust::plus<double>()) / n;
        }
        for(int i = 0; i < 4; i++)
        {
            thrust::device_vector<double> temp(n);
            thrust::transform(x.begin(), x.end(), y.begin(), temp.begin(), power_xy(i));
            _xy[i] = thrust::reduce(temp.begin(), temp.end()) / n;
        }

    }